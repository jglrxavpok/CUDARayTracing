#include "hip/hip_runtime.h"
//
// Created by jglrxavpok on 04/09/2020.
//

#include "Sphere.h"

#include <utility>
#include "Material.h"

__device__ Sphere::Sphere(Point3 center, double radius, Material* material): center(center), radius(radius), material(material) {}

__device__ void Sphere::fillResult(HitResult& result, const Ray& ray, double t) const {
    Point3 intersectionPoint = ray.at(t);
    Vec3 normal = (intersectionPoint-getCenter()) / getRadius();
    result.t = t;
    result.normal = normal;
    result.point = intersectionPoint;
    result.material = material;
}

__device__ bool Sphere::hit(const Ray &ray, double mint, double maxt, HitResult &result) const {
    auto ac = ray.origin()-getCenter();
    auto a = ray.direction().lengthSquared();
    auto halfB = dot(ac, ray.direction());
    auto c = ac.lengthSquared()-getRadius()*getRadius();
    auto discriminant = halfB * halfB - a * c;
    if(discriminant < 0.0) {
        return false;
    } else {
        double root = sqrt(discriminant);
        double solution1 = (-halfB - root) / a;
        if(solution1 > mint && solution1 < maxt) {
            fillResult(result, ray, solution1);
            return true;
        }

        double solution2 = (-halfB + root) / a;
        if(solution2 > mint && solution2 < maxt) {
            fillResult(result, ray, solution2);
            return true;
        }

        return false;
    }
}

__device__ Point3 Sphere::getCenter() const {
    return center;
}

__device__ double Sphere::getRadius() const {
    return radius;
}
