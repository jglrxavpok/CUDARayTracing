//
// Created by jglrxavpok on 05/09/2020.
//

#include "Textured.h"
#include "Lambertian.h"

__device__ Textured::Textured(Texture *texture, Material *baseMaterial, double blendFactor): texture(texture), baseMaterial(baseMaterial), blendFactor(blendFactor) {}

__device__ Textured::Textured(Texture *texture): texture(texture), blendFactor(1.0) {
    baseMaterial = new Lambertian(Color(1,1,1));
}

__device__ bool Textured::scatter(const Ray &ray, const HitResult &hit, hiprandState *rand, Color &attenuation, Ray &scattered) const {
    bool result = baseMaterial->scatter(ray, hit, rand, attenuation, scattered);
    attenuation = attenuation * (1.0-blendFactor) + blendFactor * texture->at(hit.uvwMapping);
    return result;
}
