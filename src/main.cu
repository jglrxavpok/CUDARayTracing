#include "hip/hip_runtime.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include "cudautils.h"
#include <intersectables/Sphere.h>
#include <intersectables/IntersectableGroup.h>
#include "stb_image_write.h"
#include "rt.h"
#include "Camera.h"
#include "Material.h"
#include "materials/Lambertian.h"
#include "materials/Metal.h"
#include "materials/Dielectric.h"

using std::shared_ptr;
using std::vector;

static constexpr double ASPECT_RATIO = 16.0/9.0;
static constexpr int IMAGE_HEIGHT = 200;
static constexpr int IMAGE_WIDTH = static_cast<int>(IMAGE_HEIGHT*ASPECT_RATIO);
static constexpr int MAX_BOUNCE = 5;

auto materialGround = std::make_shared<Lambertian>(Color(0.8, 0.8, 0.0));
auto materialCenter = std::make_shared<Lambertian>(Color(0.1, 0.2, 0.5));
auto materialLeft   = std::make_shared<Dielectric>(1.5);
auto materialRight  = std::make_shared<Metal>(Color(0.8, 0.6, 0.2), 0.0);

IntersectableGroup group {
    std::make_shared<Sphere>(Point3( 0.0, -100.5, -1.0), 100.0, materialGround),
    std::make_shared<Sphere>(Point3( 0.0,    0.0, -1.0), 0.5, materialCenter),
//    std::make_shared<Sphere>(Point3(-1.0,    0.0, -1.0), 0.5, materialLeft),
//    std::make_shared<Sphere>(Point3(-1.0,    0.0, -1.0), -0.45, materialLeft),
    std::make_shared<Sphere>(Point3( 1.0,    0.0, -1.0), 0.5, materialRight),
};

__device__ Color trace(const Ray& r, /*const Intersectable& world,*/ int remainingRays = MAX_BOUNCE) {
    Color skyBlue = Color(0.5, 0.7, 1.0);
    Color white = Color(1.0, 1.0, 1.0);
    Color red = Color(1.0, 0.0, 0.0);
    Color black = Color(0.0, 0.0, 0.0);

    if(remainingRays <= 0) {
        return black;
    }

    /*HitResult result{};
    if(world.hit(r, 0.001, INFINITY, result)) { // 0.001 to remove shadow acne
        Ray scattered{};
        Color attenuation{};
        if(result.material->scatter(r, result, attenuation, scattered)) {
            return attenuation * trace(scattered, world, remainingRays-1);
        }
        return white;
    }
    */

    Vec3 direction = r.direction().normalized();
    // map from -1..1 to 0..1
    auto alpha = (direction.y() + 1.0) / 2.0;
    return (1.0-alpha) * white + alpha * skyBlue;
}

__global__
void init(hiprandState* rngState) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if(x >= IMAGE_WIDTH || y >= IMAGE_HEIGHT)
        return;
    int yInversed = IMAGE_HEIGHT-y-1;
    size_t ptr = (yInversed*IMAGE_WIDTH+x)*4;
    hiprand_init(1984, ptr, 0, &rngState[ptr/4]);
}

__global__
void rayTrace(uint8_t* pixels, hiprandState* rngState) {
    Camera camera{Point3(-2,2,1), Point3(0,0,-1), Vec3(0,1,0), 20, ASPECT_RATIO};
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if(x >= IMAGE_WIDTH || y >= IMAGE_HEIGHT)
        return;
    int yInversed = IMAGE_HEIGHT-y-1;
    size_t ptr = (yInversed*IMAGE_WIDTH+x)*4;

    hiprandState localRandState = rngState[ptr/4];
    Color pixelColor{};
    for(int i = 0; i < Camera::SAMPLES_PER_PIXEL; i++) {
        double dx = hiprand_uniform_double(&localRandState);
        double dy = hiprand_uniform_double(&localRandState);
        double u = double(x+dx) / (IMAGE_WIDTH-1);
        double v = double(y+dy) / (IMAGE_HEIGHT-1);

        pixelColor += trace(camera.generateRay(u, v));
    }

    writeColor(pixels, ptr, pixelColor, Camera::SAMPLES_PER_PIXEL);
}

Color traceHost(const Ray& r, const Intersectable& world, int remainingRays = MAX_BOUNCE) {
    Color skyBlue = Color(0.5, 0.7, 1.0);
    Color white = Color(1.0, 1.0, 1.0);
    Color red = Color(1.0, 0.0, 0.0);
    Color black = Color(0.0, 0.0, 0.0);

    if(remainingRays <= 0) {
        return black;
    }

    HitResult result{};
    if(world.hit(r, 0.001 /* remove shadow acne */, INFINITY, result)) {
        Ray scattered{};
        Color attenuation{};
        if(result.material->scatter(r, result, attenuation, scattered)) {
            return attenuation * traceHost(scattered, world, remainingRays-1);
        }
        return white;
    }

    Vec3 direction = r.direction().normalized();
    // map from -1..1 to 0..1
    auto alpha = (direction.y() + 1.0) / 2.0;
    return (1.0-alpha) * white + alpha * skyBlue;
}

int main()
{
    uint8_t* pixels;
    checkCudaErrors(hipMallocManaged(&pixels, sizeof(uint8_t)*IMAGE_WIDTH*IMAGE_HEIGHT*4));

    int tileWidth = 8;
    int tileHeight = 8;
    dim3 blocks(IMAGE_WIDTH/tileWidth+1,IMAGE_HEIGHT/tileHeight+1);
    dim3 threads(tileWidth,tileHeight);

    hiprandState *deviceRNG;
    checkCudaErrors(hipMalloc((void **)&deviceRNG, IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(hiprandState)));

    init<<<blocks, threads>>>(deviceRNG);
    checkCudaErrors(hipDeviceSynchronize());

    rayTrace<<<blocks, threads>>>(pixels, deviceRNG);
    checkCudaErrors(hipDeviceSynchronize());

    stbi_write_png("./test.png", IMAGE_WIDTH, IMAGE_HEIGHT, 4, pixels, IMAGE_WIDTH*4);
    checkCudaErrors(hipFree(pixels));
    return 0;
}